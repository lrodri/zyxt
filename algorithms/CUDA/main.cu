#include "hip/hip_runtime.h"
#include <fstream>
#include <string>
#include <iostream>
#include <stdint.h>
#include <math.h>
#include <utility>
#include <sys/time.h>
#include <limits>
#include <stdlib.h>
#include <unistd.h>
#include <sstream>
#include <map>

#include <bitset>
using namespace std;

#define STEPSIZE 1 // step size in pixels, e.g. 2 = every second pixel

// compile with:
// make && ./main s11e121.txt 0 0

struct Point{
    short row;
    short col;
};

struct Pixel{
    int elevation;
    int mountingHeight;
};

struct Terrain{
    short nRows;
    short nCols;
    float cellsize;
    float xllcorner;
    float yllcorner;
    double noDataValue;
    Point *towerLocations;
    Pixel *gridTerrian;
};

clock_t start = 0, endt;
double elapsed;

void Print_Time() {
  endt = clock();
  elapsed = ((double)(endt - start)) / CLOCKS_PER_SEC;
  start = endt;

  cerr << "GPU Time: " << elapsed << endl;
}

#define CUDA_CALL(cuda_function, ...)  { \
    hipError_t status = cuda_function(__VA_ARGS__); \
    cudaEnsureSuccess(status, #cuda_function, true, __FILE__, __LINE__); \
}

bool cudaEnsureSuccess(hipError_t status, const char* status_context_description,
        bool die_on_error, const char* filename, unsigned line_number) {
    if (status_context_description == NULL)
        status_context_description = "";
    if (status == hipSuccess) {
#if REPORT_CUDA_SUCCESS
         cerr <<  "Succeeded: " << status_context_description << std::endl << std::flush;
#endif
        return true;
    }
    const char* errorString = hipGetErrorString(status);
    cerr << "CUDA Error: ";
    if (status_context_description != NULL) {
        cerr << status_context_description << ": ";
    }
    if (errorString != NULL) {
        cerr << errorString;
    }
    else {
        cerr << "(Unknown CUDA status code " << status << ")";
    }
    if (filename != NULL) {

        cerr << filename << ":" << line_number;
    }

    cerr << std::endl;
    if(die_on_error) {
        exit(EXIT_FAILURE);
            // ... or cerr << "FATAL ERROR" << etc. etc.
    }
    return false;
}

void printCudaMemory(char* info) {
    size_t free_byte ;
    size_t total_byte ;
    hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
    if ( hipSuccess != cuda_status ){
        printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
        exit(1);
    }

    double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;
    printf("[%s]GPU memory usage: used = %f, free = %f MB, total = %f MB\n", info,
        used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}

string remove_multiple_spaces(string input){
    string output;
    int k = 0, i = 0;
    for(i = 0,k=0; i < input.length(); i++, k++){
        if(input[i] == ' ' && input[i+1] == ' '){
            k--;
            continue;
        }
        output += input[i];
    }
    output += '\0';
    return output;
}

string breakStringByspace(string input){
    bool spaceCome = false;
    string output;
    for(int i = 0; i < input.length(); i++){
        if(spaceCome){
            output += input[i];
        }
        if(input[i] == ' '){
            spaceCome =  true;
        }
    }
    output += '\0';
    return output;
}

void splitStringBySpace(string input, int * output, int nCols){
    string elev = "";
    int elevationindex = 0;
    int startCol = 0;
    // ignore space at beginning of line (happens when exporting file from GRASS)
    if (input[0] == ' ') {
        startCol = 1;
    }

    for(int i = startCol; i < input.length(); i++){
        if(input[i] == ' '){
            output[elevationindex] = atoi(elev.c_str());
            elevationindex++;
            elev = "";
            continue;
        }
        elev += input[i];
        if(i == input.length() - 1){
            output[elevationindex] = atoi(elev.c_str());
        }
    }
}

void setBitHost(unsigned char *A, unsigned long long int k)
{
  A[k/8] |= 1 << (k%8);
}

__device__
void setBit(unsigned char *A, unsigned long long int k)
{
  A[k/8] |= 1 << (k%8);
}

short getBit(unsigned char *A, unsigned long long int k)
{
  return ( (A[k/8] & (1 << (k%8) )) != 0 ) ;
}

void readFileAndReturnTerrain(Terrain *h_terrain, char* file_name, int towerHeight, char* pointHeighMappingFilePath){
    ifstream fin;
    fin.open(file_name);
    // assume the file is in AAIGrid format
    string line_ncols, line_nrows, line_xllcorner, line_yllcorner, line_dx,line_dy, line_nodata;
    getline(fin, line_ncols);
    getline(fin, line_nrows);
    getline(fin, line_xllcorner);
    getline(fin, line_yllcorner);
    getline(fin, line_dx);
    getline(fin, line_dy);
    getline(fin, line_nodata);
    line_ncols = remove_multiple_spaces(line_ncols);
    line_nrows = remove_multiple_spaces(line_nrows);
    if(line_nodata.find("NODATA_value") != string::npos){
        line_nodata = remove_multiple_spaces(line_nodata);
    }
    int nCols = atoi(breakStringByspace(line_ncols).c_str());
    int nRows = atoi(breakStringByspace(line_nrows).c_str());
    float xllcorner = atof(breakStringByspace(line_xllcorner).c_str());
    float yllcorner = atof(breakStringByspace(line_yllcorner).c_str());

    // for easier calculations later on, make sure we are dealing only with an even grid size (ignore the last row and column)
    h_terrain->nCols = nCols; //% 2 == 0 ? nCols : nCols-1;
    h_terrain->nRows = nRows; //% 2 == 0 ? nRows : nRows-1;
    h_terrain->xllcorner = xllcorner;
    h_terrain->yllcorner = yllcorner;
    if(line_nodata.find("NODATA_value") == string::npos){
        h_terrain->noDataValue = 65535;
    }else{
        h_terrain->noDataValue = atoi(breakStringByspace(line_nodata).c_str());
    }
    h_terrain->gridTerrian = (Pixel*) malloc((h_terrain->nRows * h_terrain->nCols) * sizeof(Pixel));
    int rowIndex = 0;
    int * pixelInRow = (int*) malloc(h_terrain->nCols * sizeof(int));
    // loop through the file and break when finish or nRow-1 is reached
    if(line_nodata.find("NODATA_value") == string::npos){
        splitStringBySpace(line_nodata, pixelInRow, h_terrain->nCols);
        for(int j = 0; j < h_terrain->nCols; j++){
            h_terrain->gridTerrian[(h_terrain->nCols * rowIndex) + j].elevation = pixelInRow[j];
        }
        rowIndex++;
    }
    while(getline(fin, line_ncols) && rowIndex != h_terrain->nRows){
        splitStringBySpace(line_ncols, pixelInRow, h_terrain->nCols);
        for(int j = 0; j < h_terrain->nCols; j++){
            h_terrain->gridTerrian[(h_terrain->nCols * rowIndex) + j].elevation = pixelInRow[j];
        }
        rowIndex++;
    }
    free(pixelInRow);

    /*
     * Print the grid for testing
     *
     for(int i = 0; i < h_terrain->nRows; i++){
    	for(int j = 0; j < h_terrain->nCols; j++){
    		cout << h_terrain->gridTerrian[i * h_terrain->nRows + j].elevation << " ";
    	}
    	cout << endl;
      }
      exit(1);
    *
    *
    */
    /////
    ifstream pointHeighMappingFile(pointHeighMappingFilePath);
    map<long, int> towerHeightMap;
    string line;long towerIndex; int height;
    while (pointHeighMappingFile >> line){
        stringstream ss(line);
        string token;
        getline(ss, token, ',');
        towerIndex = stol(token);
        getline(ss, token, ',');
        height = stoi(token);
        towerHeightMap[towerIndex] = height;
    }
    pointHeighMappingFile.close();
    map<long, int>::iterator it;
    for(it = towerHeightMap.begin(); it != towerHeightMap.end(); ++it){
        std::cout << it->first << " => " << it->second << '\n';
    }

    /////
    h_terrain->towerLocations = (Point*) malloc((h_terrain->nRows * h_terrain->nCols)  * sizeof(Point));
    towerIndex = 0;

    for(int i = 0; i < h_terrain->nRows; i++){
        for(int j = 0; j < h_terrain->nCols; j++){
            // cout << h_terrain->gridTerrian[h_terrain->nCols * i + j].elevation << " ";
            if(towerHeightMap.count((h_terrain->nCols * i) + j) > 0){
                h_terrain->gridTerrian[(h_terrain->nCols * i) + j].mountingHeight = towerHeightMap[(h_terrain->nCols * i) + j];
            }else{
                h_terrain->gridTerrian[(h_terrain->nCols * i) + j].mountingHeight = towerHeight;
            }
            h_terrain->towerLocations[towerIndex].row = i;
            h_terrain->towerLocations[towerIndex].col = j;
            towerIndex++;
        }
        // cout << endl;
    }

    //testing of mounting height
    // for(int i = 0; i < h_terrain->nRows; i++){
    //     for(int j = 0; j < h_terrain->nCols; j++){
    //         cout << "Height of tower " << (h_terrain->nCols * i) + j << " is: " << h_terrain->gridTerrian[(h_terrain->nCols * i) + j].mountingHeight << endl;
    //     }
    // }


    //For testing tower position
    //cout << h_terrain->towerLocations[7].row << h_terrain->towerLocations[7].col << endl;
    //cout << h_terrain->gridTerrian[(h_terrain->nCols *  h_terrain->towerLocations[7].row) + h_terrain->towerLocations[7].col].elevation

    fin.close();

}

__device__
double calculateGradientOnLine(Pixel *inputTerrain, Point *observer, Point *target, int width){
    double distanceFromObserverToTarget = sqrtf(powf( (float) (target->row - observer->row) , 2.0) + powf( (float) (target->col - observer->col) , 2.0));
    int gridIndexTarget = (width * target->row) + target->col;
    int gridIndexObserver = (width * observer->row) + observer->col;
    double targetTotalElevation = inputTerrain[gridIndexTarget].elevation + inputTerrain[gridIndexTarget].mountingHeight;
    double observerTotalElevation = inputTerrain[gridIndexObserver].elevation + inputTerrain[gridIndexObserver].mountingHeight;
    return (targetTotalElevation - observerTotalElevation) / distanceFromObserverToTarget;
}

__device__
double calculateGradientOnLineWithoutTowerHeight(Pixel *inputTerrain, Point *observer, Point *target, int width){
    double distanceFromObserverToTarget = sqrtf(powf( (float) (target->row - observer->row) , 2.0) + powf( (float) (target->col - observer->col) , 2.0));
    int gridIndexTarget = (width * target->row) + target->col;
    int gridIndexObserver = (width * observer->row) + observer->col;
    double targetTotalElevation = inputTerrain[gridIndexTarget].elevation;
    double observerTotalElevation = inputTerrain[gridIndexObserver].elevation + inputTerrain[gridIndexObserver].mountingHeight;
    return (targetTotalElevation - observerTotalElevation) / distanceFromObserverToTarget;
}

int getBoundaryAroundObserver(int nRows, int nCols, Point *observerBoundary){

    int topRow = 0;
    int bottomRow = nRows-1;
    int leftCol = 0;
    int rhtCol = nCols-1;

    //storing boundary points around observer
    int observerBoundaryIndex = 0;
    for(int i = leftCol; i <= rhtCol; i = i + STEPSIZE ){
        Point p;
        p.row = topRow;
        p.col = i;
        observerBoundary[observerBoundaryIndex] = p;
        observerBoundaryIndex++;
    }
    for(int i = topRow+1; i <= bottomRow; i = i + STEPSIZE){
        Point p;
        p.row = i;
        p.col = rhtCol;
        observerBoundary[observerBoundaryIndex] = p;
        observerBoundaryIndex++;
    }
    for(int i = rhtCol-1; i >= leftCol; i = i - STEPSIZE){
        Point p;
        p.row = bottomRow;
        p.col = i;
        observerBoundary[observerBoundaryIndex] = p;
        observerBoundaryIndex++;
    }
    for(int i = bottomRow-1; i > topRow; i = i - STEPSIZE){
        Point p;
        p.row = i;
        p.col = leftCol;
        observerBoundary[observerBoundaryIndex] = p;
        observerBoundaryIndex++;
    }
    return observerBoundaryIndex;
}

__device__
void my_swap(float &x, float &y){
	float temp = 0.0;
	temp = x;
	x = y;
	y = temp;
}

__device__
int getPointsOnLine(Point *start, Point *end, Point *allPointsInLine, short nCols){
	// Bresenham's line algorithm
	 float x1 = start->row, y1 = start->col, x2 = end->row, y2 = end->col; //CHECK: these values are actually short
	  const bool steep = (fabs(y2 - y1) > fabs(x2 - x1));
	  if(steep)
	  {
		  my_swap(x1, y1);
		  my_swap(x2, y2);
	  }

	  if(x1 > x2)
	  {
		  my_swap(x1, x2);
		  my_swap(y1, y2);
	  }

	  const float dx = x2 - x1;
	  const float dy = fabs(y2 - y1);

	  float error = dx / 2.0f;
	  const int ystep = (y1 < y2) ? 1 : -1;
	  int y = (int)y1;

	  const int maxX = (int)x2;
	  int steps = 0;
	  for(int x=(int)x1; x<maxX; x++)
	  {
	    if(steep)
	    {
	    	allPointsInLine[steps].row = (int)y;
	    	allPointsInLine[steps].col = (int)x;
	    }
	    else
	    {
	    	allPointsInLine[steps].row = (int)x;
	    	allPointsInLine[steps].col = (int)y;
	    }
	    error -= dy;
	    if(error < 0)
	    {
	        y += ystep;
	        error += dx;
	    }
	    steps++;
	  }
	  return steps;
}

__global__
void calculateViewshed(Terrain terrain, unsigned char* d_viewshed, Point *viewshedBoundary, int boundarySize, int size, int iter, int r, Point* d_pointsOnLine, short sizeOfLine){

    unsigned long long int observerIndex = blockDim.x * blockIdx.x + threadIdx.x;
    // unsigned long long int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
    observerIndex = observerIndex + iter;

    if (observerIndex >= iter + r){
        // printf("exiting %llu\n", observerIndex);
        return;
    }
    // printf("%llu\n", observerIndex);


    unsigned long long int viewshedIndex = terrain.nRows * terrain.nCols * (observerIndex - iter);
    Point observerPoint = terrain.towerLocations[observerIndex];

    //skip no data values
    if(terrain.gridTerrian[observerPoint.row * terrain.nCols + observerPoint.col].elevation == terrain.noDataValue){
    	return;
    }

    for(int i = 0; i < boundarySize; i++){
         int stepsOnLine = getPointsOnLine(&observerPoint, &viewshedBoundary[i], &d_pointsOnLine[(observerIndex - iter) * sizeOfLine], terrain.nCols);
         double maxGradientBetweenObserverAndTarget = PTRDIFF_MIN;

    	//for(int j = stepsOnLine - 1; j >= 0; j--){
        for(int j = 1; j < stepsOnLine; j++){
    		unsigned long long int grid_index = (terrain.nCols * d_pointsOnLine[(observerIndex - iter) * sizeOfLine +j].row) + d_pointsOnLine[(observerIndex - iter) * sizeOfLine +j].col;
    		if(terrain.gridTerrian[grid_index].elevation != terrain.noDataValue){
    		    double gradient = calculateGradientOnLine(terrain.gridTerrian, &observerPoint, &d_pointsOnLine[(observerIndex - iter) * sizeOfLine +j], terrain.nCols);
                double gradientWithoutTowerHeight = calculateGradientOnLineWithoutTowerHeight(terrain.gridTerrian, &observerPoint, &d_pointsOnLine[(observerIndex - iter) * sizeOfLine +j], terrain.nCols);
                // if(observerIndex == 0 && i == 19){
                //     printf("%d,%f,%f\n",j, gradient, gradientWithoutTowerHeight);
                // }
                if(gradient >= maxGradientBetweenObserverAndTarget){
    		    	setBit(d_viewshed, viewshedIndex + grid_index);

    		        maxGradientBetweenObserverAndTarget = gradientWithoutTowerHeight;
                    // maxGradientBetweenObserverAndTarget = gradient;
    		    }
    		}
    	}
         //calculating for the last point
        unsigned long long int grid_index = (terrain.nCols * viewshedBoundary[i].row) + viewshedBoundary[i].col;
        if(terrain.gridTerrian[grid_index].elevation != terrain.noDataValue){
            double gradient = calculateGradientOnLine(terrain.gridTerrian, &observerPoint, &viewshedBoundary[i], terrain.nCols);
            double gradientWithoutTowerHeight = calculateGradientOnLineWithoutTowerHeight(terrain.gridTerrian, &observerPoint, &viewshedBoundary[i], terrain.nCols);
            if(gradient >= maxGradientBetweenObserverAndTarget){
                setBit(d_viewshed, viewshedIndex + grid_index);
                maxGradientBetweenObserverAndTarget = gradientWithoutTowerHeight;
            }
        }

    }

}

int getIndexOfTower(Point *towerLocations, int size, int row, int col){
    int index = -1;
    for(int i = 0; i < size; i++){
        if(towerLocations[i].row == row && towerLocations[i].col == col){
            index = i;
        }
    }
    return index;
}

int main(int argc, char* argv[]){
    int dev = 1;
    CUDA_CALL(hipSetDevice, dev);
    hipError_t err = hipSuccess;
    Terrain h_terrain;
    Pixel *d_pixel_grid, *h_pixel_grid; Point *d_towers, *h_towers;

    if(argc < 2){
        cout << "Not enough arguments!" << endl;
        cout << "example: ./main /path/to/file" << endl;
        return 1;
    }
    cout << "Reading Input File...." << endl;
    readFileAndReturnTerrain(&h_terrain, argv[1], atoi(argv[3]), argv[5]);
    //exit(1);
    cout << "After file output" << endl;
    h_pixel_grid = h_terrain.gridTerrian;
    h_towers = h_terrain.towerLocations;

    int THREADS = atoi(argv[2]); //4

    unsigned long long int totalThreads = THREADS;
    cout << "totalThreads: " << totalThreads << endl;
    cout << "nRows: " << h_terrain.nRows << ", nCols: " << h_terrain.nCols << endl;
    float lenGlobal = (float)(h_terrain.nRows*h_terrain.nCols)/8;
    std::cout << "MemLength " << lenGlobal<< std::endl;
    printCudaMemory((char*)"1");
    CUDA_CALL(hipMalloc, (void**)&d_pixel_grid, (h_terrain.nRows * h_terrain.nCols) * sizeof(Pixel));
    CUDA_CALL(hipMemcpy, d_pixel_grid, h_pixel_grid, (h_terrain.nRows * h_terrain.nCols) * sizeof(Pixel), hipMemcpyHostToDevice);
    printCudaMemory((char*)"2");
    CUDA_CALL(hipMalloc, (void**)&d_towers, totalThreads * sizeof(Point));
    CUDA_CALL(hipMemcpy, d_towers, h_towers, totalThreads * sizeof(Point), hipMemcpyHostToDevice);
    printCudaMemory((char*)"3");
    h_terrain.towerLocations = d_towers;
    h_terrain.gridTerrian = d_pixel_grid;

    int boundarySize = (h_terrain.nCols + h_terrain.nRows) / STEPSIZE;
    Point *h_viewshedBoundary, *d_viewshedBoundary;
    h_viewshedBoundary = (Point*) malloc( ((boundarySize) * 2) * sizeof(Point));
    int totalBoundarySize = getBoundaryAroundObserver(h_terrain.nRows, h_terrain.nCols, h_viewshedBoundary);
    CUDA_CALL(hipMalloc, (void**)&d_viewshedBoundary, ((boundarySize) * 2) * sizeof(Point));
    CUDA_CALL(hipMemcpy, d_viewshedBoundary, h_viewshedBoundary, ((boundarySize ) * 2) * sizeof(Point), hipMemcpyHostToDevice);
    printCudaMemory((char*)"4");
    cout << "Total Boundary Size: " << totalBoundarySize << endl;
    cout << "Starting Kernel with # of threads " << totalThreads<<endl;
    size_t heap;
    CUDA_CALL(hipDeviceGetLimit, &heap, hipLimitMallocHeapSize);
    cout << "Heap size before = " << heap << endl;
    // this is dirty fix
    // CUDA_CALL(hipDeviceSetLimit, hipLimitMallocHeapSize, heap*max(1,THREADS/1000));
    // CUDA_CALL(hipDeviceGetLimit, &heap, hipLimitMallocHeapSize);
    cout << "Heap size after = " << heap << endl;

    hipDeviceProp_t myCUDA;
    if (hipGetDeviceProperties(&myCUDA, dev) == hipSuccess)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %zdByte; compute v%d.%d; clock: %d kHz\n",
            myCUDA.name, myCUDA.totalGlobalMem, (int)myCUDA.major,
            (int)myCUDA.minor, (int)myCUDA.clockRate);
    }

    int threadsPerBlock = myCUDA.maxThreadsPerBlock;
    int blocksPerGrid = (totalThreads + threadsPerBlock - 1) / threadsPerBlock;
    cout << "Maximum threads per block = " << threadsPerBlock << endl;
    cout << "Blocks per Grid = " << blocksPerGrid << endl;
    cout << "Size of global viewshed" << ceil(totalThreads * lenGlobal *sizeof(unsigned char)) << endl;
    unsigned char *g_viewshed = (unsigned char *) calloc(ceil(totalThreads * lenGlobal *sizeof(unsigned char)), sizeof(unsigned char));
    if (g_viewshed == NULL)
    {
        std::cout << "Memory allocation failed" << std::endl;
        exit(1);
    }
    unsigned long long int MAX_ELEMENTS = 250000UL * 250000UL; //25600000000; // 500x500

    int r = max(int(MAX_ELEMENTS / totalThreads), 1); //make sure we process at least 1 row
    printf("Rows to process in each iteration %d\n", r);
    printf("Iterations %f\n", ceil(totalThreads * totalThreads / MAX_ELEMENTS));

    for (int i = 0; i < totalThreads; i+=r) { //step

        float len = (float)(totalThreads)/8;
        printf("Start Iteration %d, %f , %f\n", i/r,len,ceil(r * len *sizeof(unsigned char)) );
        // std::cout << r << " " << len << " " << ceil(r * len *sizeof(unsigned char)) << std::endl;

        // we are in the last thread, only process the remaining viewsheds
        if (i+r > totalThreads) {
            r = totalThreads - i;
            // std::cout << "Remainder " << r << std::endl;
        }
        unsigned char *h_viewshed, *d_viewshed;

        // round up to the next full byte
        h_viewshed = (unsigned char *) calloc(ceil(r * len *sizeof(unsigned char)), sizeof(unsigned char));
        // cout << "viewshed size " << r * len *sizeof(unsigned char) * 8 << " elements --> allocated memory: " << ceil(r * len *sizeof(unsigned char)) << " bytes" << endl;
        // std::cout << "MemLength~~ " <<ceil(r * len *sizeof(unsigned char)) << std::endl;
        CUDA_CALL(hipMalloc, (unsigned char**)&d_viewshed, (ceil(r * len *sizeof(unsigned char))));
        CUDA_CALL(hipMemcpy, d_viewshed, h_viewshed, ceil(r* len *sizeof(unsigned char)), hipMemcpyHostToDevice);
        printCudaMemory((char*)"Viewshed");

        // allocate memory for the max number of possible points, i.e. map size
        int sizeOfLine = sqrt(pow(h_terrain.nRows , 2.0) + pow(h_terrain.nCols, 2.0) );

        Point *h_pointsOnLine,*d_pointsOnLine;
        h_pointsOnLine = (Point*)malloc(r * sizeOfLine * sizeof(Point)); //FIXME
        CUDA_CALL(hipMalloc, (Point **)&d_pointsOnLine, (r * sizeOfLine * sizeof(Point)));
        CUDA_CALL(hipMemcpy, d_pointsOnLine, h_pointsOnLine, r* sizeOfLine *sizeof(Point), hipMemcpyHostToDevice);
        printCudaMemory((char*)"Viewshed");

        std::cout << "Iteration Start" << std::endl;
        Print_Time();

        // calculateViewshed<<<blocksPerGrid, threadsPerBlock>>>(h_terrain, d_viewshed, d_viewshedBoundary, totalBoundarySize, totalThreads, i, r);
        calculateViewshed<<<blocksPerGrid, threadsPerBlock>>>(h_terrain, d_viewshed, d_viewshedBoundary, totalBoundarySize, totalThreads, i, r, d_pointsOnLine, sizeOfLine);

        hipError_t errSync  = hipGetLastError();
        if (errSync != hipSuccess) {
            printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
            exit(EXIT_FAILURE);
        }
        cout << "Waiting for all jobs to finish..." << endl;
        hipError_t errAsync = hipDeviceSynchronize();
        if (errAsync != hipSuccess) {
            printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
            exit(EXIT_FAILURE);
        }

        std::cout << "Iteration stop" << std::endl;
        Print_Time();

        cout << "Copy viewsheds from device to host" << endl;
        err = hipMemcpy(h_viewshed, d_viewshed, ceil(r* len *sizeof(unsigned char)), hipMemcpyDeviceToHost);
        if (err != hipSuccess){
            cout << "Failed to copy viewsheds grid from device to host. Error String " << hipGetErrorString(err) << err << endl;
            exit(EXIT_FAILURE);
        }
        cout << "Viewshed calculation complete" << endl;
        // Print_Time();

        // memcpy(g_viewshed + (r * len * sizeof(unsigned char)), h_viewshed, ceil(r* len *sizeof(unsigned char)));

        // exit(1);

         // commenting it due to seg fault after iteration 1
        unsigned long long int count = 0;
        for(unsigned long long int k = (i * totalThreads); k < ( (unsigned long long int)(i * totalThreads) + (r * totalThreads) ); k++){
            // std::cout << k << " " <<getBit(h_viewshed, count)<< std::endl;
            // g_viewshed.set(k, getBit(h_viewshed, count));
            if(getBit(h_viewshed, count)){
                // std::cout << k << std::endl;
                setBitHost(g_viewshed,k);
            }
            count++;
            // g_viewshed[k] = getBit(h_viewshed, j * (h_terrain.nRows * h_terrain.nCols) + k)
        }


        // FILE *fp;
        // char buff[10];
        // sprintf(buff,"%d",i);
        // fp = fopen(buff, "wb");
        // // long tRows = (h_terrain.nRows * h_terrain.nCols);
        // for(int j = r-1; j < r; j++){
        //     for(int k = 0; k < totalThreads; k++){
        //         // unsigned long long int index = ((h_terrain.nRows * h_terrain.nCols) * (unsigned long long int) indexOfTower) + ((h_terrain.nCols * j) + k);
        //         unsigned long long int mainIndex = j * totalThreads + k;
        //         if(getBit(h_viewshed, mainIndex)){
        //             // cout << "1 " << mainIndex << endl;
        //             fprintf(fp, "1 ");
        //         }
        //         else{
        //             // cout << "0 " << mainIndex << endl;
        //             fprintf(fp, "0 ");
        //         }
        //     }
        //     fprintf(fp, "\n");
        // }
        // fclose(fp);

        free(h_viewshed);
        free(h_pointsOnLine);
        hipFree(d_pointsOnLine);
        hipFree(d_viewshed);
    }

    // std::cout << "Writing to file..." << std::endl;
    // FILE *fp;
    // fp = fopen("out.txt", "wb");

    // for(int j = 0; j < totalThreads; j++){
    //     for(int k = 0; k < totalThreads; k++){
    //         // unsigned long long int index = ((h_terrain.nRows * h_terrain.nCols) * (unsigned long long int) indexOfTower) + ((h_terrain.nCols * j) + k);
    //         if(getBit(g_viewshed, j * (h_terrain.nRows * h_terrain.nCols) + k)){
    //             fprintf(fp, "1 ");
    //         }
    //         else
    //             fprintf(fp, "0 ");
    //     }
    //     fprintf(fp, "\n");
    // }
    // fclose(fp);

    // for(int j = 0; j < totalThreads; j++){
    //         for(int k = 0; k < totalThreads; k++){
    //             // unsigned long long int index = ((h_terrain.nRows * h_terrain.nCols) * (unsigned long long int) indexOfTower) + ((h_terrain.nCols * j) + k);
    //             if(getBit(g_viewshed, j * (h_terrain.nRows * h_terrain.nCols) + k)){
    //                 cout << "1 ";
    //             }
    //             else{
    //                 cout << "0 ";
    //             }
    //         }
    //         cout << endl;
    //     }
    std::cout << "Writing to binary file..." << std::endl;
    FILE * write_ptr;
    write_ptr = fopen(argv[4],"wb");
    fwrite(g_viewshed, sizeof(unsigned char), ceil(totalThreads * lenGlobal), write_ptr);
    fclose(write_ptr);

    free(g_viewshed);

    hipFree(d_pixel_grid);
    hipFree(d_towers);
    hipFree(d_viewshedBoundary);
    free(h_pixel_grid);
    free(h_towers);
    free(h_viewshedBoundary);
    // reset the device
    err = hipDeviceReset();
    return 0;
}
